
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

#define N 100


__global__ void fast_sum(int* a, int* b, int* c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;
    for (int i = 0; i < N; ++i) {
        a[i] = i;
        b[i] = N - i;
    }

    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    fast_sum<<<1, N>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    for (int i = 0; i < N; ++i) {
        std::cout << c[i] << " ";
    }


    return 0;
}

#include "hip/hip_runtime.h"
%%cuda
#include <iostream>
#include <stdio.h>


const int N = 10;
const int blocksPerGrid = 2;
const int threadsPerBlock = 4;

void fill(int* arr, int size) {
    for (int i = 0; i < size; ++i)
        *(arr + i) = i + 1;
}

__global__ void dot(int* a, int* b, int* c) {
    __shared__ int cache[threadsPerBlock];
    int cacheIndex = threadIdx.x;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    int temp = 0;
    for (int i = tid; i < N; i += stride) {
        temp += a[i] * b[i];
    }
    cache[cacheIndex] = temp;
    __syncthreads();
    
    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}

int main() {
    int a[N], b[N], partial_c[blocksPerGrid];
    int *dev_a, *dev_b, *dev_partial_c;
    fill(a, N);
    fill(b, N);

    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_partial_c, blocksPerGrid * sizeof(int));

    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    dot<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_partial_c);

    hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_partial_c);

    int dot_product = 0;
    for (int i = 0; i < blocksPerGrid; ++i)
        dot_product += partial_c[i];

    std::cout << dot_product;
        
    return 0;
}

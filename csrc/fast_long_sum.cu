
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

#define N 10


void fill(int* arr, int size) {
    for (int i = 0; i < size; ++i)
        *(arr + i) = i + 1;
}

__global__ void sum(int* a, int* b, int* c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        printf("Thread ID after: %d\n", tid);
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
        printf("Thread ID before: %d\n", tid);
    }
}

int main() {
    int a[N], b[N], c[N];
    int* dev_a, *dev_b, *dev_c;
    fill(a, N);
    fill(b, N);

    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice);

    sum<<<2, 2>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i)
        std::cout << c[i] << " ";     
    
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
